#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <memory.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <>

#include "chacha20_shared.cuh"
#include "../../_utils/conversion_utils.cuh"

void chacha20_process_file(const char* input_path, const char* output_path, const uint8_t* key) {
    clock_t c_start = clock();

    FILE* input_file = fopen(input_path, "rb");
    FILE* output_file = fopen(output_path, "wb");

    if (input_file == NULL) {
        fprintf(stderr, "Error: Could not open input file %s\n", input_path);
        return;
    }

    if (output_file == NULL) {
        fprintf(stderr, "Error: Could not open output file %s\n", output_path);
        return;
    }

    // Initialize the context
    uint32_t h_init_state[16];
    uint8_t nonce[8] = { 0 };
    chacha20_init(h_init_state, key, nonce);

    // Get the file size
    fseek(input_file, 0, SEEK_END);
    long file_size = ftell(input_file);
    fseek(input_file, 0, SEEK_SET);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    clock_t c_file_read = 0;
    clock_t c_file_write = 0;

    uint32_t *d_init_state;
    hipMalloc((uint32_t**)&d_init_state, sizeof(h_init_state));
    hipMemcpy(d_init_state, &h_init_state, sizeof(h_init_state), hipMemcpyHostToDevice);

    uint64_t start_counter = 0;
    while (file_size > 0)
    {
        unsigned long long size_to_process = MIN(file_size, deviceProp.totalGlobalMem / 3);

        // Store the file in a buffer
        clock_t c_file_read_start = clock();
        uint8_t* h_buffer = (uint8_t*)malloc(size_to_process);
        if (h_buffer == NULL) {
            fprintf(stderr, "Error: Could not allocate memory for the file\n");
            exit(EXIT_FAILURE);
        }
        size_t bytes_read = fread(h_buffer, 1, size_to_process, input_file);
        clock_t c_file_read_end = clock();
        c_file_read += c_file_read_end - c_file_read_start;

        uint8_t* d_buffer;
        hipMalloc((uint8_t**)&d_buffer, bytes_read);
        hipMemcpy(d_buffer, h_buffer, bytes_read, hipMemcpyHostToDevice);

        // Determine the number of blocks
        size_t num_chacha20_blocks = bytes_read / sizeof(h_init_state);
        if (bytes_read % sizeof(h_init_state) != 0) {
            num_chacha20_blocks++;
        }
        size_t num_threads_per_block = MIN(256, num_chacha20_blocks);
        size_t num_blocks = num_chacha20_blocks / num_threads_per_block;
        if (num_chacha20_blocks % num_threads_per_block != 0) {
            num_blocks++;
        }

        // Encrypt the file
        chacha20_process << <num_blocks, num_threads_per_block >> > (d_init_state, d_buffer, d_buffer, bytes_read, start_counter);
        hipDeviceSynchronize();

        // Handle errors
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
            exit(EXIT_FAILURE);
        }

        // Copy the encrypted data back to the host
        hipMemcpy(h_buffer, d_buffer, bytes_read, hipMemcpyDeviceToHost);

        hipFree(d_buffer);

        // Append the encrypted data to the output file
        clock_t c_file_write_start = clock();
        fwrite(h_buffer, 1, bytes_read, output_file);
        clock_t c_file_write_end = clock();
		c_file_write += c_file_write_end - c_file_write_start;

        free(h_buffer);

        file_size -= size_to_process;
        start_counter += num_chacha20_blocks;
    }
    // Free the memory on the device
	hipFree(d_init_state);

    fclose(input_file);
    fclose(output_file);

    clock_t c_end = clock();
    clock_t c_total = c_end - c_start;

    printf("File read: %f ms\n", (double)c_file_read / CLOCKS_PER_SEC * 1000);
    printf("File write: %f ms\n", (double)c_file_write / CLOCKS_PER_SEC * 1000);
    printf("Total: %f ms\n", (double)c_total / CLOCKS_PER_SEC * 1000);
}

__host__ void chacha20_init(uint32_t init_state[16], const uint8_t* key, const uint8_t nonce[8]) {
    const char constants[17] = "expand 32-byte k";

    init_state[0] = LE(constants + 0);
    init_state[1] = LE(constants + 4);
    init_state[2] = LE(constants + 8);
    init_state[3] = LE(constants + 12);
    init_state[4] = LE(key + 0);
    init_state[5] = LE(key + 4);
    init_state[6] = LE(key + 8);
    init_state[7] = LE(key + 12);
    init_state[8] = LE(key + 16);
    init_state[9] = LE(key + 20);
    init_state[10] = LE(key + 24);
    init_state[11] = LE(key + 28);
    init_state[12] = 0; // Counter
    init_state[13] = 0; // Counter
    init_state[14] = LE(nonce + 0);
    init_state[15] = LE(nonce + 4);
}

__global__ void chacha20_process(uint32_t* init_state, uint8_t* in, uint8_t* out, size_t size_to_encrypt, uint64_t start_counter)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ uint32_t shared_state[16];
    if (threadIdx.x == 0)
	{
        memcpy(shared_state, init_state, sizeof(shared_state));
	}
    __syncthreads();

    uint32_t stream[16];

    size_t block_idx_start = thread_id * sizeof(stream);
    size_t block_size = MIN(sizeof(stream), size_to_encrypt - block_idx_start);

    chacha20_block(shared_state, stream, start_counter + thread_id);
    chacha20_xor((uint8_t*)stream, in + block_idx_start, out + block_idx_start, block_size);
}

__device__ void chacha20_block(uint32_t init_state[16], uint32_t output[16], uint64_t counter)
{
    memcpy(output, init_state, 64); // Do not use "sizeof(init_state)" as CUDA will consider it as 8 instead of 64
    chacha20_set_counter(output, counter);

    int i = 10;

    while (i--)
    {
        CHACHA20_QUARTER_ROUND(output, 0, 4, 8, 12)
            CHACHA20_QUARTER_ROUND(output, 1, 5, 9, 13)
            CHACHA20_QUARTER_ROUND(output, 2, 6, 10, 14)
            CHACHA20_QUARTER_ROUND(output, 3, 7, 11, 15)
            CHACHA20_QUARTER_ROUND(output, 0, 5, 10, 15)
            CHACHA20_QUARTER_ROUND(output, 1, 6, 11, 12)
            CHACHA20_QUARTER_ROUND(output, 2, 7, 8, 13)
            CHACHA20_QUARTER_ROUND(output, 3, 4, 9, 14)
    }
    for (i = 0; i < 16; ++i)
    {
        uint32_t result = output[i] + init_state[i];
        FROM_LE((uint8_t*)(output + i), result);
    }
}

__device__ inline void chacha20_xor(uint8_t* keystream, uint8_t* in, uint8_t* out, size_t length)
{
    for (size_t i = 0; i < length; i++)
	{
		out[i] = in[i] ^ keystream[i];
	}
}

__device__ void chacha20_set_counter(uint32_t* state, uint64_t counter)
{
    state[12] = counter & UINT32_C(0xFFFFFFFF);
    state[13] = (counter >> 32);
}