#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <memory.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <>

#include "chacha20.cuh"
#include "../../_utils/conversion_utils.cuh"

#define NB_STREAMS 6

void chacha20_process_file(const char* input_path, const char* output_path, const uint8_t* key) {
    clock_t c_start = clock();

    FILE* input_file = fopen(input_path, "rb");
    FILE* output_file = fopen(output_path, "wb");

    if (input_file == NULL) {
        fprintf(stderr, "Error: Could not open input file %s\n", input_path);
        return;
    }

    if (output_file == NULL) {
        fprintf(stderr, "Error: Could not open output file %s\n", output_path);
        return;
    }

    // Initialize the context
    chacha20_ctx h_ctx;
    uint8_t nonce[8] = { 0 };
    chacha20_init(&h_ctx, key, nonce);

    // Get the file size
    fseek(input_file, 0, SEEK_END);
    long file_size = ftell(input_file);
    fseek(input_file, 0, SEEK_SET);

    unsigned long long chacha20_count_blocks = file_size / sizeof(h_ctx.init_state);
    if (chacha20_count_blocks < NB_STREAMS) {
		fprintf(stderr, "Error: The file is too small to be encrypted using this implementation of streams\n");
		return;
	}
    size_t count_blocks_to_process_per_stream = chacha20_count_blocks / NB_STREAMS;

    chacha20_ctx* d_ctx;
    hipMalloc((chacha20_ctx**)&d_ctx, sizeof(chacha20_ctx));
    hipMemcpy(d_ctx, &h_ctx, sizeof(chacha20_ctx), hipMemcpyHostToDevice);

    uint64_t start_counter = 0;
    hipStream_t streams[NB_STREAMS] = { 0 };
    uint8_t* d_buffers[NB_STREAMS] = { 0 };
    size_t buffer_size[NB_STREAMS] = { 0 };
    uint8_t count_streams = 0;
    while (file_size > 0)
    {
        size_t size_to_process = count_blocks_to_process_per_stream * sizeof(h_ctx.init_state);
        if (count_streams == NB_STREAMS - 1)
		{
			size_to_process = file_size;
		}

        hipStream_t stream;
        hipStreamCreate(&stream);

        uint8_t* d_buffer;
        hipMalloc((uint8_t**)&d_buffer, size_to_process);

        // Store the part of the file to process in a buffer
        uint8_t* h_buffer = (uint8_t*)malloc(size_to_process);
        if (h_buffer == NULL) {
            fprintf(stderr, "Error: Could not allocate memory for the file\n");
            exit(EXIT_FAILURE);
        }
        size_t bytes_read = fread(h_buffer, 1, size_to_process, input_file);

		hipMemcpyAsync(d_buffer, h_buffer, bytes_read, hipMemcpyHostToDevice, stream);

        // Determine the number of blocks for this kernel call
        size_t num_chacha20_blocks = bytes_read / sizeof(h_ctx.init_state);
        if (bytes_read % sizeof(h_ctx.init_state) != 0) {
            num_chacha20_blocks++;
        }
        size_t num_threads_per_block = MIN(256, num_chacha20_blocks);
        size_t num_blocks = num_chacha20_blocks / num_threads_per_block;
        if (num_chacha20_blocks % num_threads_per_block != 0) {
            num_blocks++;
        }

        // Encrypt the file
        chacha20_process<<<num_blocks, num_threads_per_block, 0, stream>>> (d_ctx, d_buffer, d_buffer, bytes_read, start_counter);

        free(h_buffer);

        file_size -= bytes_read;
        start_counter += num_chacha20_blocks;

        streams[count_streams] = stream;
        d_buffers[count_streams] = d_buffer;
        buffer_size[count_streams] = bytes_read;
        count_streams++;
    }

    hipDeviceSynchronize();

    for (uint8_t i = 0; i < count_streams; i++)
	{
        hipStream_t stream = streams[i];
        size_t size_to_process = buffer_size[i];
        uint8_t* d_buffer = d_buffers[i];
        uint8_t* h_buffer = (uint8_t*)malloc(size_to_process);

		hipMemcpyAsync(h_buffer, d_buffer, size_to_process, hipMemcpyDeviceToHost, stream);
		fwrite(h_buffer, 1, size_to_process, output_file);

		free(h_buffer);
		hipFree(d_buffer);
		hipStreamDestroy(streams[i]);
	}

    // Free the memory on the device
	hipFree(d_ctx);

    fclose(input_file);
    fclose(output_file);

    clock_t c_end = clock();
    clock_t c_total = c_end - c_start;

    printf("Total: %f ms\n", (double)c_total / CLOCKS_PER_SEC * 1000);
}

__host__ void chacha20_init(chacha20_ctx* ctx, const uint8_t* key, const uint8_t nonce[8]) {
    const char constants[17] = "expand 32-byte k";

    ctx->init_state[0] = LE(constants + 0);
    ctx->init_state[1] = LE(constants + 4);
    ctx->init_state[2] = LE(constants + 8);
    ctx->init_state[3] = LE(constants + 12);
    ctx->init_state[4] = LE(key + 0);
    ctx->init_state[5] = LE(key + 4);
    ctx->init_state[6] = LE(key + 8);
    ctx->init_state[7] = LE(key + 12);
    ctx->init_state[8] = LE(key + 16);
    ctx->init_state[9] = LE(key + 20);
    ctx->init_state[10] = LE(key + 24);
    ctx->init_state[11] = LE(key + 28);
    ctx->init_state[12] = 0; // Counter
    ctx->init_state[13] = 0; // Counter
    ctx->init_state[14] = LE(nonce + 0);
    ctx->init_state[15] = LE(nonce + 4);
}

__global__ void chacha20_process(chacha20_ctx* ctx, uint8_t* in, uint8_t* out, size_t size_to_encrypt, uint64_t start_counter)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t stream[16];

    size_t block_idx_start = thread_id * sizeof(stream);
    size_t block_size = MIN(sizeof(stream), size_to_encrypt - block_idx_start);

    chacha20_block(ctx, stream, start_counter + thread_id);
    chacha20_xor((uint8_t*)stream, in + block_idx_start, out + block_idx_start, block_size);
}

__device__ void chacha20_block(chacha20_ctx* ctx, uint32_t output[16], uint64_t counter)
{
    memcpy(output, ctx->init_state, sizeof(ctx->init_state));
    chacha20_set_counter(output, counter);

    int i = 10;

    while (i--)
    {
        CHACHA20_QUARTER_ROUND(output, 0, 4, 8, 12)
            CHACHA20_QUARTER_ROUND(output, 1, 5, 9, 13)
            CHACHA20_QUARTER_ROUND(output, 2, 6, 10, 14)
            CHACHA20_QUARTER_ROUND(output, 3, 7, 11, 15)
            CHACHA20_QUARTER_ROUND(output, 0, 5, 10, 15)
            CHACHA20_QUARTER_ROUND(output, 1, 6, 11, 12)
            CHACHA20_QUARTER_ROUND(output, 2, 7, 8, 13)
            CHACHA20_QUARTER_ROUND(output, 3, 4, 9, 14)
    }
    for (i = 0; i < 16; ++i)
    {
        uint32_t result = output[i] + ctx->init_state[i];
        FROM_LE((uint8_t*)(output + i), result);
    }
}

__device__ inline void chacha20_xor(uint8_t* keystream, uint8_t* in, uint8_t* out, size_t length)
{
    for (size_t i = 0; i < length; i++)
	{
        uint8_t result = in[i] ^ keystream[i];
		out[i] = result;
    }
}

__device__ void chacha20_set_counter(uint32_t* state, uint64_t counter)
{
    state[12] = counter & UINT32_C(0xFFFFFFFF);
    state[13] = (counter >> 32);
}